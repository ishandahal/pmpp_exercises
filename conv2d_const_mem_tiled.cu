#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>

#define RADIUS 3
#define FILTER_DIM (2 * RADIUS + 1)
#define INPUT_TILE_DIM 16 // *** Block dim needs to match this ***
#define OUTPUT_TILE_DIM ((INPUT_TILE_DIM) - 2 * RADIUS)

__constant__ float F[FILTER_DIM][FILTER_DIM];

__global__ void conv2d_tiled_kernel(float *N, float *P, int radius, int width,
                                    int height) {
  int row = blockIdx.y * OUTPUT_TILE_DIM + threadIdx.y - radius;
  int col = blockIdx.x * OUTPUT_TILE_DIM + threadIdx.x - radius;

  __shared__ float Nds[INPUT_TILE_DIM][INPUT_TILE_DIM];
  // Load input tile after checking the boundary condition
  if (row >= 0 && row < height && col >= 0 && col < width) {
    Nds[threadIdx.y][threadIdx.x] = N[row * width + col];
  } else {
    Nds[threadIdx.y][threadIdx.x] = 0.0f;
  }
  __syncthreads();

  int tileRow = threadIdx.y - radius;
  int tileCol = threadIdx.x - radius;
  if (row >= 0 && row < height && col >= 0 && col < width) {
    if (tileRow >= 0 && tileRow < OUTPUT_TILE_DIM && tileCol >= 0 &&
        tileCol < OUTPUT_TILE_DIM) {
      float Pvalue = 0.0f;
      for (int fRow = 0; fRow < FILTER_DIM; ++fRow) {
        for (int fCol = 0; fCol < FILTER_DIM; ++fCol) {
          Pvalue += F[fRow][fCol] * Nds[fRow + tileRow][fCol + tileCol];
        }
      }
      P[row * width + col] = Pvalue;
    }
  }
}

// Helper function to print a small matrix
void printMatrix(float *matrix, int width, int height) {
  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) {
      std::cout << matrix[y * width + x] << " ";
    }
    std::cout << std::endl;
  }
}

// CPU implementation of 2D convolution with index clamping
void cpuConvolution2D(float *input, float *output, float *kernel, int width,
                      int height, int kernelWidth, int kernelHeight) {
  int radius = (kernelWidth - 1) / 2;
  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) {
      float sum = 0.0f;
      for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
          int inputX = x + kx;
          int inputY = y + ky;
          int kernelIdx = (ky + radius) * kernelWidth + (kx + radius);
          if (inputX >= 0 && inputX < width && inputY >= 0 && inputY < height) {
            sum += input[inputY * width + inputX] * kernel[kernelIdx];
          }
        }
      }
      output[y * width + x] = sum;
    }
  }
}

int main() {
  // Define dimensions
  const int width = 1500;
  const int height = 1300;
  const int radius = RADIUS;
  const int kernelWidth = FILTER_DIM;
  const int kernelHeight = FILTER_DIM;
  const int outputWidth = width;
  const int outputHeight = height;

  // Calculate sizes
  size_t inputSize = width * height * sizeof(float);
  size_t kernelSize = kernelWidth * kernelHeight * sizeof(float);
  size_t outputSize = outputWidth * outputHeight * sizeof(float);

  // Host memory allocation
  float *h_input = (float *)malloc(inputSize);
  float *h_kernel = (float *)malloc(kernelSize);
  float *h_output_gpu = (float *)malloc(outputSize);
  float *h_output_cpu = (float *)malloc(outputSize);

  // Initialize input and kernel with random values
  srand(time(NULL));
  for (int i = 0; i < width * height; i++) {
    h_input[i] = static_cast<float>(rand()) / RAND_MAX;
  }
  for (int i = 0; i < kernelWidth * kernelHeight; i++) {
    h_kernel[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // Device memory allocation
  float *d_input, *d_kernel, *d_output;
  hipMalloc(&d_input, inputSize);
  hipMalloc(&d_kernel, kernelSize);
  hipMalloc(&d_output, outputSize);

  // Copy data to device
  hipMemcpy(d_input, h_input, inputSize, hipMemcpyHostToDevice);
  // cudaMemcpy(d_kernel, h_kernel, kernelSize, cudaMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(F), h_kernel, kernelWidth * kernelHeight * sizeof(float));

  // Define block and grid dimensions
  dim3 blockDim(INPUT_TILE_DIM, INPUT_TILE_DIM);
  dim3 gridDim((outputWidth + OUTPUT_TILE_DIM - 1) /
                   OUTPUT_TILE_DIM, // Ceiling division by OUTPUT_TILE_DIM
               (outputHeight + OUTPUT_TILE_DIM - 1) / OUTPUT_TILE_DIM);
  // Launch kernel

  conv2d_tiled_kernel<<<gridDim, blockDim>>>(d_input, d_output, radius, width,
                                             height);

  // Check for kernel launch errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Kernel launch failed: " << hipGetErrorString(err)
              << std::endl;
    return 1;
  }

  // Synchronize device
  hipDeviceSynchronize();

  // Copy GPU results back to host
  hipMemcpy(h_output_gpu, d_output, outputSize, hipMemcpyDeviceToHost);

  // Compute CPU reference result
  cpuConvolution2D(h_input, h_output_cpu, h_kernel, width, height, kernelWidth,
                   kernelHeight);

  // Compare GPU and CPU results
  const float tolerance =
      1e-5f; // Small tolerance for floating-point differences
  bool match = true;
  for (int i = 0; i < outputWidth * outputHeight; i++) {
    float diff = std::abs(h_output_gpu[i] - h_output_cpu[i]);
    if (diff > tolerance) {
      match = false;
      std::cout << "Mismatch at index " << i << " (y=" << i / width
                << ", x=" << i % width << "): " << "GPU=" << h_output_gpu[i]
                << ", CPU=" << h_output_cpu[i] << ", diff=" << diff
                << std::endl;
    }
  }

  if (match) {
    std::cout << "GPU and CPU results match within tolerance!" << std::endl;
  } else {
    std::cout << "GPU and CPU results differ!" << std::endl;
  }
  /*
  // Optional: Print samples
  std::cout << "\nInput sample:" << std::endl;
  printMatrix(h_input, width, std::min(5, height));
  std::cout << "\nKernel:" << std::endl;
  printMatrix(h_kernel, kernelWidth, kernelHeight);
  std::cout << "\nGPU Output sample:" << std::endl;
  printMatrix(h_output_gpu, outputWidth, std::min(5, outputHeight));
  std::cout << "\nCPU Output sample:" << std::endl;
  printMatrix(h_output_cpu, outputWidth, std::min(5, outputHeight));
  */

  // Clean up
  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_output);
  free(h_input);
  free(h_kernel);
  free(h_output_gpu);
  free(h_output_cpu);

  return 0;
}
